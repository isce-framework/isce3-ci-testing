#include "hip/hip_runtime.h"
#include <cstdio>
#include <string>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#include "gpuSignal.h"
#include <isce3/cuda/except/Error.h>

#define THRD_PER_BLOCK 1024 // Number of threads per block (should always %32==0)

namespace isce3::cuda::signal {

/** copy left half columns of lo-res to left most columns of hi-res
*  copy right half columns of lo-res to right most columns of hi-res
*  0 fill columns in hi-res that has nothing copied to it
*  e.g. lo-res 2x4 -> hi-res 2x6
*       1 1 1 1       1 1 0 0 1 1
*       1 1 1 1       1 1 0 0 1 1
*  @param[out] pointer to hi res data
*  @param[in] pointer to lo res data
*  @param[in] number of rows
*  @param[in] number of lo res columns
*  @param[in] number of hi res columns
*/
template<class T>
__global__ void rangeShift_g(thrust::complex<T>*data_hi_res,
                             const thrust::complex<T>* __restrict__ data_lo_res,
                             int n_rows,
                             int n_cols_lo,
                             int n_cols_hi)
{
    // determine 1-d index of hi-res array
    const auto i_hi = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;

    if (i_hi < n_cols_hi * n_rows) {
        // determine row and column from 1-d index
        auto i_row = i_hi / n_cols_hi;
        auto i_col = i_hi % n_cols_hi;

        // check if lo column is on the left
        if (i_col < (n_cols_lo + 1)/ 2) {
            auto i_lo = i_row * n_cols_lo + i_col;
            data_hi_res[i_hi] = data_lo_res[i_lo];
        // check if lo column is on the right
        } else if (i_col > n_cols_hi - n_cols_lo / 2) {
            auto i_lo = i_row * n_cols_lo + i_col - (n_cols_hi - n_cols_lo);
            data_hi_res[i_hi] = data_lo_res[i_lo];
        // else zero in the middle
        } else {
            data_hi_res[i_hi] = thrust::complex<T>(0.0, 0.0);
        }
    }
}

/** apply complex shift impact
*  @param[in] pointer to data to be multiplied by shift impact
*  @param[in] pointer to shift impact data
*  @param[in] number of total elements to be multiplied
*/
template<class T>
__global__ void shiftImpact_g(thrust::complex<T> *data,
                              const thrust::complex<T>* __restrict__ shift_impact,
                              size_t n_elements)
{
    const auto i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;

    if (i < n_elements) {
        data[i] *= shift_impact[i];
    }
}

/** normalize in-place on device
*  @param[in] pointer to data to be normalized
*  @param[in] normalization factor
*  @param[in] number of total elements to be normalized
*/
template<class T>
__global__ void normalize_g(thrust::complex<T> *data,
                            const T normalization,
                            size_t n_elements)
{
    const auto i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;

    if (i < n_elements) {
        data[i] /= normalization;
    }
}


/** Constructor **/
template<class T>
gpuSignal<T>::
gpuSignal(hipfftType _type) {
    _cufft_type = _type;
    _plan_set = false;
    _d_data = NULL;
    _d_data_set = false;
}


/** Destructor **/
template<class T>
gpuSignal<T>::
~gpuSignal() {
    if (_plan_set) {
        hipfftDestroy(_plan);
    }

    if (_d_data_set) {
        checkCudaErrors(hipFree(_d_data));
    }
}


/** sets up range 1D FFT
*  @param[in] ncolumns number of columns of the block of data
*  @param[in] nrows number of rows of the block of data
 */
template<class T>
void gpuSignal<T>::
rangeFFT(int ncolumns, int nrows)
{
    _configureRangeFFT(ncolumns, nrows);

    fftPlan(_rank, _n, _howmany,
            _inembed, _istride, _idist,
            _onembed, _ostride, _odist);
}


/** sets up azimuth 1D FFT
*  @param[in] ncolumns number of columns of the block of data
*  @param[in] nrows number of rows of the block of data
 */
template<class T>
void gpuSignal<T>::
azimuthFFT(int ncolumns, int nrows)
{
    _configureAzimuthFFT(ncolumns, nrows);

    fftPlan(_rank, _n, _howmany,
            _inembed, _istride, _idist,
            _onembed, _ostride, _odist);
}


/** sets up 2D FFT
*  @param[in] ncolumns number of columns of the block of data
*  @param[in] nrows number of rows of the block of data
 */
template<class T>
void gpuSignal<T>::
FFT2D(int ncolumns, int nrows)
{
    _n_elements = nrows * ncolumns;
    if (_plan_set) {
        hipfftDestroy(_plan);
    }

    checkCudaErrors(hipfftCreate(&_plan));
    _plan_set = true;
    size_t worksize;
    checkCudaErrors(hipfftMakePlan2d(_plan, nrows, ncolumns, _cufft_type, &worksize));
}


/** sets up 1D cufft
*  @param[in] rank rank of the transform (1: for one dimensional and 2: for two dimensional transform)
*  @param[in] size size of each transform (ncols: for range FFT, nrows: for azimuth FFT)
*  @param[in] howmany number of FFT transforms for a block of data (nrows: for range FFT, ncols: for azimuth FFT)
*  @param[in] inembed
*  @param[in] istride
*  @param[in] idist
*  @param[in] onembed
*  @param[in] ostride
*  @param[in] odist
*/
template <class T>
void gpuSignal<T>::
fftPlan(int rank, int *n, int howmany,
        int *inembed, int istride, int idist,
        int *onembed, int ostride, int odist)
{
    if (_plan_set) {
        hipfftDestroy(_plan);
    }

    checkCudaErrors(hipfftCreate(&_plan));
    _plan_set = true;
    size_t worksize;
    checkCudaErrors(hipfftMakePlanMany(_plan, rank, n,
                                      inembed, istride, idist,
                                      onembed, ostride, odist,
                                      _cufft_type, _howmany, &worksize));
}


/** finds next power of 2 >= given size
*   @param[in] N the actual length of a signal
*   @param[in] fftLength next power of two
*/
template <class T>
void gpuSignal<T>::
nextPowerOfTwo(size_t N, size_t &fftLength)
{
    for (size_t i = 0; i < 17; ++i) {
        fftLength = std::pow(2, i);
        if (fftLength >= N) {
            break;
        }
    }
}


/** sets range specific FFT parameters
*   @param[in] ncolumns number of columns
*   @param[in] nrows number of rows
*/
template <class T>
void gpuSignal<T>::
_configureRangeFFT(int ncolumns, int nrows)
{
    _rank = 1;
    _n[0] = ncolumns;
    _n[1] = 0;

    _howmany = nrows;

    _inembed[0] = ncolumns;
    _inembed[0] = 0;

    _istride = 1;
    _idist = ncolumns;

    _onembed[0] = ncolumns;
    _onembed[1] = 0;

    _ostride = 1;
    _odist = ncolumns;

    _n_elements = nrows * ncolumns;

    _rows = nrows;
    _columns = ncolumns;
}


/** sets up azimuth specific FFT parameters
*   @param[in] ncolumns number of columns
*   @param[in] nrows number of rows
*/
template <class T>
void gpuSignal<T>::
_configureAzimuthFFT(int ncolumns, int nrows)
{
    _rank = 1;
    _n[0] = nrows;
    _n[1] = 0;

    _howmany = ncolumns;

    _inembed[0] = nrows;
    _inembed[1] = 0;

    _istride = ncolumns;
    _idist = 1;

    _onembed[0] = nrows;
    _onembed[1] = 0;

    _ostride = ncolumns;
    _odist = 1;

    _n_elements = nrows * ncolumns;

    _rows = nrows;
    _columns = ncolumns;
}


/** copies data from host to device
*   @param[in] pointer to host data
*/
template<class T>
void gpuSignal<T>::
dataToDevice(std::complex<T> *input)
{
    if (!_d_data_set) {
        size_t input_size = _n_elements*sizeof(thrust::complex<T>);
        // allocate input
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&_d_data), input_size));
        // copy input
        checkCudaErrors(hipMemcpy(_d_data, input, input_size, hipMemcpyHostToDevice));
        _d_data_set = true;
    }
}


/** copies data from host to device
*   @param[in] valarray on host data
*/
template<class T>
void gpuSignal<T>::
dataToDevice(std::valarray<std::complex<T>> &input)
{
    if (!_d_data_set) {
        size_t input_size = input.size()*sizeof(thrust::complex<T>);
        // allocate input
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&_d_data), input_size));
        // copy input
        checkCudaErrors(hipMemcpy(_d_data, &input[0], input_size, hipMemcpyHostToDevice));
        _d_data_set = true;
    }
}

template<class T>
void gpuSignal<T>::
dataToHost(std::complex<T> *output)
{
    if (_d_data_set) {
        size_t output_size = _n_elements*sizeof(thrust::complex<T>);
        // copy output
        checkCudaErrors(hipMemcpy(output, _d_data, output_size, hipMemcpyDeviceToHost));
    }
}

template<class T>
void gpuSignal<T>::
dataToHost(std::valarray<std::complex<T>> &output)
{
    if (_d_data_set) {
        size_t output_size = _n_elements*sizeof(thrust::complex<T>);
        // copy output
        checkCudaErrors(hipMemcpy(&output[0], _d_data, output_size, hipMemcpyDeviceToHost));
    }
}


/** unnormalized forward complex float transform performed in place on class data
*/
template<class T>
void gpuSignal<T>::
forward()
{
    if (_plan_set && _d_data_set) {
        forwardDevMem(_d_data);
    }
}


/** unnormalized forward complex float transform performed on given device data
*   @param[in] pointer to source data on device
*   @param[out] pointer to output data on device
*/
template<>
void gpuSignal<float>::
forwardDevMem(thrust::complex<float> *input, thrust::complex<float> *output)
{
    // transform
    if (_plan_set) {
        hipfftComplex* _input = reinterpret_cast<hipfftComplex*>(input);
        hipfftComplex* _output = reinterpret_cast<hipfftComplex*>(output);
        checkCudaErrors(hipfftExecC2C(_plan, _input, _output, HIPFFT_FORWARD));
        checkCudaErrors(hipDeviceSynchronize());
    }
}


/** unnormalized forward complex double transform performed on given device data
*   @param[in] pointer to source data on device
*   @param[out] pointer to output data on device
*/
template<>
void gpuSignal<double>::
forwardDevMem(thrust::complex<double> *input, thrust::complex<double> *output)
{
    // transform
    if (_plan_set) {
        hipfftDoubleComplex* _input = reinterpret_cast<hipfftDoubleComplex*>(input);
        hipfftDoubleComplex* _output = reinterpret_cast<hipfftDoubleComplex*>(output);
        checkCudaErrors(hipfftExecZ2Z(_plan, _input, _output, HIPFFT_FORWARD));
        checkCudaErrors(hipDeviceSynchronize());
    }
}


/** unnormalized forward complex transform performed in place on given device data
*   @param[in] pointer to source/output data on device
*/
template<class T>
void gpuSignal<T>::
forwardDevMem(thrust::complex<T> *dataInPlace) {
    forwardDevMem(dataInPlace, dataInPlace);
}


/** unnormalized forward complex float transform
*  @param[in] pointer to input block of data
*  @param[in] pointer to output block of spectrum
*/
template<class T>
void gpuSignal<T>::
forwardC2C(std::complex<T> *input, std::complex<T> *output)
{
    if (_plan_set) {
        size_t input_size = _n_elements*sizeof(std::complex<T>);
        size_t output_size = _n_elements*sizeof(std::complex<T>);

        // allocate device memory
        T *d_input;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
        T *d_output;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

        // copy input
        checkCudaErrors(hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice));

        // transform
        checkCudaErrors(hipfftExecC2C(_plan, reinterpret_cast<hipfftComplex *>(d_input),
                                    reinterpret_cast<hipfftComplex *>(d_input),
                                    HIPFFT_FORWARD));

        checkCudaErrors(hipDeviceSynchronize());

        // copy output
        checkCudaErrors(hipMemcpy(output, d_input, input_size, hipMemcpyDeviceToHost));

        checkCudaErrors(hipFree(d_input));
        checkCudaErrors(hipFree(d_output));
    }
}


/** unnormalized forward complex float transform
*  @param[in] valarray containing input block of data
*  @param[in] valarray containing output block of spectrum
*/
template<class T>
void gpuSignal<T>::
forwardC2C(std::valarray<std::complex<T>> &input, std::valarray<std::complex<T>> &output)
{
    if (_plan_set) {
        size_t input_size = input.size()*sizeof(std::complex<T>);
        size_t output_size = output.size()*sizeof(std::complex<T>);

        // allocate device memory
        T *d_input;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
        T *d_output;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

        // copy input
        checkCudaErrors(hipMemcpy(d_input, &input[0], input_size, hipMemcpyHostToDevice));

        // transform
        checkCudaErrors(hipfftExecC2C(_plan, reinterpret_cast<hipfftComplex *>(d_input),
                                    reinterpret_cast<hipfftComplex *>(d_input),
                                    HIPFFT_FORWARD));

        checkCudaErrors(hipDeviceSynchronize());

        // copy output
        checkCudaErrors(hipMemcpy(&output[0], d_input, input_size, hipMemcpyDeviceToHost));

        checkCudaErrors(hipFree(d_input));
        checkCudaErrors(hipFree(d_output));
    }
}


/** unnormalized forward complex double transform
*  @param[in] pointer to input block of data
*  @param[in] pointer to output block of spectrum
*/
template<class T>
void gpuSignal<T>::
forwardZ2Z(std::complex<T> *input, std::complex<T> *output)
{
    if (_plan_set) {
        size_t input_size = _n_elements*sizeof(std::complex<T>);
        size_t output_size = _n_elements*sizeof(std::complex<T>);

        // allocate device memory
        T *d_input;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
        T *d_output;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

        // copy input
        checkCudaErrors(hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice));

        // transform
        checkCudaErrors(hipfftExecZ2Z(_plan, reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                    reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                    HIPFFT_FORWARD));

        checkCudaErrors(hipDeviceSynchronize());

        // copy output
        checkCudaErrors(hipMemcpy(output, d_input, input_size, hipMemcpyDeviceToHost));

        checkCudaErrors(hipFree(d_input));
        checkCudaErrors(hipFree(d_output));
    }
}


/** unnormalized forward complex double transform
*  @param[in] valarray containing input block of data
*  @param[in] valarray containing output block of spectrum
*/
template<class T>
void gpuSignal<T>::
forwardZ2Z(std::valarray<std::complex<T>> &input, std::valarray<std::complex<T>> &output)
{
    if (_plan_set) {
        size_t input_size = input.size()*sizeof(std::complex<T>);
        size_t output_size = output.size()*sizeof(std::complex<T>);

        // allocate device memory
        T *d_input;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
        T *d_output;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

        // copy input
        checkCudaErrors(hipMemcpy(d_input, &input[0], input_size, hipMemcpyHostToDevice));

        // transform
        checkCudaErrors(hipfftExecZ2Z(_plan, reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                    reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                    HIPFFT_FORWARD));

        checkCudaErrors(hipDeviceSynchronize());

        // copy output
        checkCudaErrors(hipMemcpy(&output[0], d_input, input_size, hipMemcpyDeviceToHost));

        checkCudaErrors(hipFree(d_input));
        checkCudaErrors(hipFree(d_output));
    }
}


/** unnormalized forward double to complex double transform
*  @param[in] pointer to input block of data
*  @param[in] pointer to output block of spectrum
*/
template<class T>
void gpuSignal<T>::
forwardD2Z(T *input, std::complex<T> *output)
{
    if (_plan_set) {
        size_t input_size = _n_elements*sizeof(T);
        size_t output_size = _n_elements*sizeof(std::complex<T>);

        // allocate device memory
        T *d_input;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
        T *d_output;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

        // copy input
        checkCudaErrors(hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice));

        // transform (implicitly forward)
        checkCudaErrors(hipfftExecD2Z(_plan, reinterpret_cast<hipfftDoubleReal *>(d_input),
                                    reinterpret_cast<hipfftDoubleComplex *>(d_output)));

        checkCudaErrors(hipDeviceSynchronize());

        // copy output
        checkCudaErrors(hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost));

        checkCudaErrors(hipFree(d_input));
        checkCudaErrors(hipFree(d_output));
    }
}


/** unnormalized forward complex float transform
*  @param[in] pointer to input block of data
*  @param[in] pointer to output block of spectrum
*/
template<>
void gpuSignal<float>::
forward(std::complex<float> *input, std::complex<float> *output)
{
    forwardC2C(input, output);
}


/** unnormalized forward complex double transform
*  @param[in] pointer to input block of data
*  @param[in] pointer to output block of spectrum
*/
template<>
void gpuSignal<double>::
forward(std::complex<double> *input, std::complex<double> *output)
{
    forwardZ2Z(input, output);
}


/** unnormalized forward complex float transform
*  @param[in] valarray containing input block of data
*  @param[in] valarray containing output block of spectrum
*/
template<>
void gpuSignal<float>::
forward(std::valarray<std::complex<float>> &input, std::valarray<std::complex<float>> &output)
{
    forwardC2C(input, output);
}


/** unnormalized forward complex double transform
*  @param[in] valarray containing input block of data
*  @param[in] valarray containing output block of spectrum
*/
template<>
void gpuSignal<double>::
forward(std::valarray<std::complex<double>> &input, std::valarray<std::complex<double>> &output)
{
    forwardZ2Z(input, output);
}


/** unnormalized inverse complex float transform performed in place on class data
*/
template<class T>
void gpuSignal<T>::
inverse()
{
    if (_plan_set && _d_data_set) {
        inverseDevMem(_d_data);
    }
}


/** unnormalized inverse complex float transform performed on given device data
*   @param[in] pointer to source data on device
*   @param[out] pointer to output data on device
*/
template<>
void gpuSignal<float>::
inverseDevMem(thrust::complex<float> *input, thrust::complex<float> *output)
{
    // transform
    if (_plan_set) {
        hipfftComplex* _input = reinterpret_cast<hipfftComplex*>(input);
        hipfftComplex* _output = reinterpret_cast<hipfftComplex*>(output);
        checkCudaErrors(hipfftExecC2C(_plan, _input, _output, HIPFFT_BACKWARD));
        checkCudaErrors(hipDeviceSynchronize());
    }
}


/** unnormalized inverse complex double transform performed on given device data
*   @param[in] pointer to source data on device
*   @param[out] pointer to output data on device
*/
template<>
void gpuSignal<double>::
inverseDevMem(thrust::complex<double> *input, thrust::complex<double> *output)
{
    // transform
    if (_plan_set) {
        hipfftDoubleComplex* _input = reinterpret_cast<hipfftDoubleComplex*>(input);
        hipfftDoubleComplex* _output = reinterpret_cast<hipfftDoubleComplex*>(output);
        checkCudaErrors(hipfftExecZ2Z(_plan, _input, _output, HIPFFT_BACKWARD));
        checkCudaErrors(hipDeviceSynchronize());
    }
}


/** unnormalized inverse complex transform performed in place on given device data
*   @param[in] pointer to source/output data on device
*/
template<class T>
void gpuSignal<T>::
inverseDevMem(thrust::complex<T> *dataInPlace) {
    inverseDevMem(dataInPlace, dataInPlace);
}


/** unnormalized inverse complex float transform
*  @param[in] pointer to input block of spectrum
*  @param[in] pointer to output block of data
*/
template<class T>
void gpuSignal<T>::
inverseC2C(std::complex<T> *input, std::complex<T> *output)
{
    if (_plan_set) {
        size_t input_size = _n_elements*sizeof(std::complex<T>);
        size_t output_size = _n_elements*sizeof(std::complex<T>);

        // allocate device memory
        T *d_input;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
        T *d_output;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

        // copy input
        checkCudaErrors(hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice));

        // transform
        checkCudaErrors(hipfftExecC2C(_plan, reinterpret_cast<hipfftComplex *>(d_input),
                                    reinterpret_cast<hipfftComplex *>(d_input),
                                    HIPFFT_BACKWARD));

        checkCudaErrors(hipDeviceSynchronize());

        // copy output
        checkCudaErrors(hipMemcpy(output, d_input, input_size, hipMemcpyDeviceToHost));

        checkCudaErrors(hipFree(d_input));
        checkCudaErrors(hipFree(d_output));
    }

}


/** unnormalized inverse complex float transform
*  @param[in] valarray containing input block of spectrum
*  @param[in] valarray containing output block of data
*/
template<class T>
void gpuSignal<T>::
inverseC2C(std::valarray<std::complex<T>> &input, std::valarray<std::complex<T>> &output)
{
    if (_plan_set) {
        size_t input_size = input.size()*sizeof(std::complex<T>);
        size_t output_size = output.size()*sizeof(std::complex<T>);

        // allocate device memory
        T *d_input;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
        T *d_output;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

        // copy input
        checkCudaErrors(hipMemcpy(d_input, &input[0], input_size, hipMemcpyHostToDevice));

        // transform
        checkCudaErrors(hipfftExecC2C(_plan, reinterpret_cast<hipfftComplex *>(d_input),
                                    reinterpret_cast<hipfftComplex *>(d_input),
                                    HIPFFT_BACKWARD));

        checkCudaErrors(hipDeviceSynchronize());

        // copy output
        checkCudaErrors(hipMemcpy(&output[0], d_input, input_size, hipMemcpyDeviceToHost));

        checkCudaErrors(hipFree(d_input));
        checkCudaErrors(hipFree(d_output));
    }
}


/** unnormalized inverse complex double transform
*  @param[in] pointer to input block of spectrum
*  @param[in] pointer to output block of data
*/
template<class T>
void gpuSignal<T>::
inverseZ2Z(std::complex<T> *input, std::complex<T> *output)
{
    if (_plan_set) {
        size_t input_size = _n_elements*sizeof(std::complex<T>);
        size_t output_size = _n_elements*sizeof(std::complex<T>);

        // allocate device memory
        T *d_input;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
        T *d_output;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

        // copy input
        checkCudaErrors(hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice));

        // transform
        checkCudaErrors(hipfftExecZ2Z(_plan, reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                    reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                    HIPFFT_BACKWARD));

        checkCudaErrors(hipDeviceSynchronize());

        // copy output
        checkCudaErrors(hipMemcpy(output, d_input, input_size, hipMemcpyDeviceToHost));

        checkCudaErrors(hipFree(d_input));
        checkCudaErrors(hipFree(d_output));
    }
}


/** unnormalized inverse complex double transform
*  @param[in] valarray containing input block of spectrum
*  @param[in] valarray containing output block of data
*/
template<class T>
void gpuSignal<T>::
inverseZ2Z(std::valarray<std::complex<T>> &input, std::valarray<std::complex<T>> &output)
{
    if (_plan_set) {
        size_t input_size = input.size()*sizeof(std::complex<T>);
        size_t output_size = output.size()*sizeof(std::complex<T>);

        // allocate device memory
        T *d_input;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
        T *d_output;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

        // copy input
        checkCudaErrors(hipMemcpy(d_input, &input[0], input_size, hipMemcpyHostToDevice));

        // transform
        checkCudaErrors(hipfftExecZ2Z(_plan, reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                    reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                    HIPFFT_BACKWARD));

        checkCudaErrors(hipDeviceSynchronize());

        // copy output
        checkCudaErrors(hipMemcpy(&output[0], d_input, input_size, hipMemcpyDeviceToHost));

        checkCudaErrors(hipFree(d_input));
        checkCudaErrors(hipFree(d_output));
    }
}


/** unnormalized inverse complex double to double transform
*  @param[in] pointer to input block of spectrum
*  @param[in] pointer to output block of data
*/
template<class T>
void gpuSignal<T>::
inverseZ2D(std::complex<T> *input, T *output)
{
    if (_plan_set) {
        size_t input_size = _n_elements*sizeof(std::complex<T>);
        size_t output_size = _n_elements*sizeof(T);

        // allocate device memory
        T *d_input;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
        T *d_output;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

        // copy input
        checkCudaErrors(hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice));

        // transform (implicitly inverse)
        checkCudaErrors(hipfftExecZ2D(_plan, reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                    reinterpret_cast<hipfftDoubleReal *>(d_output)));

        checkCudaErrors(hipDeviceSynchronize());

        // copy output
        checkCudaErrors(hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost));

        checkCudaErrors(hipFree(d_input));
        checkCudaErrors(hipFree(d_output));
    }
}


/** unnormalized inverse complex float transform
*  @param[in] pointer to input block of spectrum
*  @param[in] pointer to output block of data
*/
template<>
void gpuSignal<float>::
inverse(std::complex<float> *input, std::complex<float> *output)
{
    inverseC2C(input, output);
}


/** unnormalized inverse complex double transform
*  @param[in] pointer to input block of spectrum
*  @param[in] pointer to output block of data
*/
template<>
void gpuSignal<double>::
inverse(std::complex<double> *input, std::complex<double> *output)
{
    inverseZ2Z(input, output);
}


/** unnormalized inverse complex float transform
*  @param[in] valarray containing input block of spectrum
*  @param[in] valarray containing output block of data
*/
template<>
void gpuSignal<float>::
inverse(std::valarray<std::complex<float>> &input, std::valarray<std::complex<float>> &output)
{
    inverseC2C(input, output);
}


/** unnormalized inverse complex double transform
*  @param[in] valarray containing input block of spectrum
*  @param[in] valarray containing output block of data
*/
template<>
void gpuSignal<double>::
inverse(std::valarray<std::complex<double>> &input, std::valarray<std::complex<double>> &output)
{
    inverseZ2Z(input, output);
}


/** normalized complex float column/range-wise upsampling
*  @param[in] valarray containing lo res data
*  @param[in] valarray containing hi res data
*  @param[in] rows in both lo and hi res data
*  @param[in] columns in lo res data
*  @param[in] upsample factor
*  @param[in] valarray containing shift impact
*/
template<class T>
void gpuSignal<T>::
upsample(std::valarray<std::complex<T>> &input,
         std::valarray<std::complex<T>> &output,
         int row,
         int col,
         int upsampleFactor,
         std::valarray<std::complex<T>> &shiftImpact)
{
    if (_plan_set) {
        size_t input_size = _n_elements*sizeof(thrust::complex<T>);
        size_t output_size = upsampleFactor * _n_elements*sizeof(thrust::complex<T>);

        // allocate device memory
        thrust::complex<T> *d_input;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
        thrust::complex<T> *d_output;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

        // copy input
        checkCudaErrors(hipMemcpy(d_input, &input[0], input_size, hipMemcpyHostToDevice));

        // forward transform
        forwardDevMem(d_input);

        // determine block layout
        dim3 block(THRD_PER_BLOCK);
        dim3 grid((input_size+(THRD_PER_BLOCK-1))/THRD_PER_BLOCK);

        // shift data prior to upsampling transform
        rangeShift_g<<<grid, block>>>(d_output, d_input,
                                      _rows, _columns,
                                      upsampleFactor*_columns);

        if (shiftImpact.size() == output.size()) {
            thrust::complex<T> *d_shift_impact;
            size_t shift_size = shiftImpact.size()*sizeof(thrust::complex<T>);
            checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_shift_impact), shift_size));
            checkCudaErrors(hipMemcpy(d_shift_impact,
                                       &shiftImpact[0],
                                       shift_size,
                                       hipMemcpyHostToDevice));

            shiftImpact_g<<<grid, block>>>(d_output, d_shift_impact, shift_size);

            checkCudaErrors(hipFree(d_shift_impact));
        }

        // set inverse transform
        rangeFFT(upsampleFactor*col, 1);

        // inverse transformation
        inverseDevMem(d_output);

        // normalize
        normalize_g<<<grid, block>>>(d_output, static_cast<T>(_columns), _n_elements);

        // copy output
        checkCudaErrors(hipMemcpy(&output[0], d_output, output_size, hipMemcpyDeviceToHost));

        checkCudaErrors(hipFree(d_input));
        checkCudaErrors(hipFree(d_output));
    }
}


/** normalized complex column/range-wise upsampling
*  @param[in] valarray containing lo res data
*  @param[in] valarray containing hi res data
*  @param[in] rows in both lo and hi res data
*  @param[in] columns in lo res data
*  @param[in] upsample factor
*/
template<class T>
void gpuSignal<T>::
upsample(std::valarray<std::complex<T>> &input,
         std::valarray<std::complex<T>> &output,
         int row,
         int nfft,
         int upsampleFactor)
{
    std::valarray<std::complex<T>> shiftImpact(0);

    upsample(input, output,
            row,
            nfft,
            upsampleFactor,
            shiftImpact);
}

/** upsample performed on device
*  @param[in] forward signal object
*  @param[in] inverse signal object
*  @param[in] pointer to data to be upsampled
*  @param[in] pointer to upsampled data
*/
template<class T>
void upsample(gpuSignal<T> &fwd,
        gpuSignal<T> &inv,
        thrust::complex<T> *input,
        thrust::complex<T> *output)
{
    fwd.forwardDevMem(input);

    // determine block layout
    auto input_size = fwd.getNumElements();
    dim3 block(THRD_PER_BLOCK);
    dim3 grid((input_size+(THRD_PER_BLOCK-1))/THRD_PER_BLOCK);

    // shift data prior to upsampling transform
    rangeShift_g<<<grid, block>>>(output,
                                  input,
                                  fwd.getRows(),
                                  fwd.getColumns(),
                                  inv.getColumns());

    inv.inverseDevMem(output);

    // columns**2 because fwd transform not normalized
    normalize_g<<<grid, block>>>(
            output,
            static_cast<T>(inv.getColumns()*inv.getColumns()),
            inv.getNumElements());
}


/** upsample performed on device
*  @param[in] forward signal object
*  @param[in] inverse signal object
*  @param[in] pointer to data to be upsampled
*  @param[in] pointer to upsampled data
*  @param[in] pointer to shift impact data
*/
template<class T>
void upsample(gpuSignal<T> &fwd,
        gpuSignal<T> &inv,
        thrust::complex<T> *input,
        thrust::complex<T> *output,
        thrust::complex<T> *shiftImpact)
{
    fwd.forwardDevMem(input);

    // determine block layout
    dim3 block(THRD_PER_BLOCK);
    auto nInvElements = inv.getNumElements();
    dim3 grid((nInvElements + (THRD_PER_BLOCK - 1)) / THRD_PER_BLOCK);

    // shift data prior to upsampling transform
    rangeShift_g<T><<<grid, block>>>(output,
                                     input,
                                     fwd.getRows(),
                                     fwd.getColumns(),
                                     inv.getColumns());

    shiftImpact_g<T><<<grid, block>>>(output,
                                      shiftImpact,
                                      nInvElements);

    inv.inverseDevMem(output);

    // columns**2 because fwd transform not normalized
    normalize_g<T><<<grid, block>>>(output,
                                    static_cast<T>(fwd.getColumns()),
                                    nInvElements);
}


/** upsample performed on device
*  @param[in] forward signal object
*  @param[in] inverse signal object
*  @param[in] valarray containing data to be upsampled
*  @param[in] valarray containing upsampled data
*/
template<class T>
void upsample(gpuSignal<T> &fwd,
        gpuSignal<T> &inv,
        std::valarray<std::complex<T>> &input,
        std::valarray<std::complex<T>> &output)
{
    std::valarray<std::complex<T>> empty_shift(0);

    upsample(fwd,
            inv,
            input,
            output,
            empty_shift);
}


/** upsample performed on device
*  @param[in] forward signal object
*  @param[in] inverse signal object
*  @param[in] valarray containing data to be upsampled
*  @param[in] valarray containing upsampled data
*  @param[in] valarray containing shift impact data
*/
template<class T>
void upsample(gpuSignal<T> &fwd,
        gpuSignal<T> &inv,
        std::valarray<std::complex<T>> &input,
        std::valarray<std::complex<T>> &output,
        std::valarray<std::complex<T>> &shiftImpact)
{
    auto input_size = input.size()*sizeof(thrust::complex<T>);
    thrust::complex<T> *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    checkCudaErrors(hipMemcpy(d_input, &input[0], input_size, hipMemcpyHostToDevice));

    auto output_size = output.size()*sizeof(thrust::complex<T>);
    thrust::complex<T> *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));
    checkCudaErrors(hipMemcpy(d_output, &output[0], output_size, hipMemcpyHostToDevice));

    if (shiftImpact.size() > 0) {
        auto shiftImpact_size = shiftImpact.size()*sizeof(thrust::complex<T>);
        thrust::complex<T> *d_shiftImpact;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_shiftImpact), shiftImpact_size));
        checkCudaErrors(hipMemcpy(d_shiftImpact, &shiftImpact[0], shiftImpact_size, hipMemcpyHostToDevice));
        upsample(fwd,
                inv,
                d_input,
                d_output,
                d_shiftImpact);
        checkCudaErrors(hipFree(d_shiftImpact));
    } else {
        upsample(fwd,
                inv,
                d_input,
                d_output);
    }

    checkCudaErrors(hipMemcpy(&output[0], d_output, output_size, hipMemcpyDeviceToHost));

    output /= static_cast<T>(inv.getColumns());

    checkCudaErrors(hipFree(d_input));
    checkCudaErrors(hipFree(d_output));
}


/*
explicit instantiations
 */
template class gpuSignal<float>;
template class gpuSignal<double>;

template void
upsample<float>(gpuSignal<float> &fwd,
        gpuSignal<float> &inv,
        std::valarray<std::complex<float>> &input,
        std::valarray<std::complex<float>> &output);

template void
upsample<double>(gpuSignal<double> &fwd,
        gpuSignal<double> &inv,
        std::valarray<std::complex<double>> &input,
        std::valarray<std::complex<double>> &output);

template<class T>
void upsample(gpuSignal<T> &fwd,
        gpuSignal<T> &inv,
        std::valarray<std::complex<T>> &input,
        std::valarray<std::complex<T>> &output);

template void upsample<float>(gpuSignal<float> &fwd,
        gpuSignal<float> &inv,
        thrust::complex<float> *input,
        thrust::complex<float> *output,
        thrust::complex<float> *shiftImpact);

template void upsample<double>(gpuSignal<double> &fwd,
        gpuSignal<double> &inv,
        thrust::complex<double> *input,
        thrust::complex<double> *output,
        thrust::complex<double> *shiftImpact);

} // namespace isce3::cuda::signal
